#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "efficient_plus.h"

namespace StreamCompaction {
    namespace EfficientPlus {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer &timer() {
            static PerformanceTimer timer;
            return timer;
        }

        __host__ __device__ unsigned conflictFreeIndex(unsigned index) {
            // Number of banks: 32
            unsigned offset = index / 32U;
            offset += offset / 32U;
            return index + offset;
        }

        __device__ int &conflictFreeGet(int *data, unsigned index) {
            return data[conflictFreeIndex(index)];
        }

        template <unsigned ElementsPerThread>
        __global__ void scanPerBlock(unsigned n, int *__restrict__ data, int *__restrict__ sums) {
            unsigned tileSize = blockDim.x * ElementsPerThread * 2U;

            extern __shared__ int sharedData[];
            unsigned startSharedIndex = threadIdx.x * ElementsPerThread * 2U;
            unsigned endSharedIndex = startSharedIndex + ElementsPerThread * 2U;
            unsigned startGlobalIndex = blockIdx.x * tileSize + startSharedIndex;
            unsigned endGlobalIndex = min(startGlobalIndex + ElementsPerThread * 2U, n);
            {
                // Copy data to shared memory.
                unsigned sharedIndex = startSharedIndex;
                unsigned globalIndex = startGlobalIndex;
                for (; globalIndex < endGlobalIndex; ++sharedIndex, ++globalIndex) {
                    conflictFreeGet(sharedData, sharedIndex) = data[globalIndex];
                }
                for (; sharedIndex < endSharedIndex; ++sharedIndex) {
                    conflictFreeGet(sharedData, sharedIndex) = 0;
                }
            }
            __syncthreads();

            // Up-sweep
            for (unsigned offset = 1U; offset < tileSize; offset *= 2U) {
#pragma unroll
                for (unsigned i = 0U; i < ElementsPerThread; ++i) {
                    unsigned index = (threadIdx.x * ElementsPerThread + i + 1U) * offset * 2U - 1U;
                    if (index < tileSize) {
                        conflictFreeGet(sharedData, index) +=
                            conflictFreeGet(sharedData, index - offset);
                    }
                }
                __syncthreads();
            }

            if (threadIdx.x == 0U) {
                int &lastElement = conflictFreeGet(sharedData, tileSize - 1U);
                // Save the total sum of this block to the sums array.
                if (sums != nullptr) {
                    // sums may be nullptr for the last recursion.
                    sums[blockIdx.x] = lastElement;
                }
                // Clear the last element before down-sweep.
                lastElement = 0;
            }
            __syncthreads();

            // Down-sweep
            for (unsigned offset = tileSize / 2U; offset > 0U; offset /= 2U) {
#pragma unroll
                for (unsigned i = 0U; i < ElementsPerThread; ++i) {
                    unsigned index = (threadIdx.x * ElementsPerThread + i + 1U) * offset * 2U - 1U;
                    if (index < tileSize) {
                        int &leftChild = conflictFreeGet(sharedData, index - offset);
                        int &rightChild = conflictFreeGet(sharedData, index);
                        int oldLeftChild = leftChild;
                        leftChild = rightChild;
                        rightChild += oldLeftChild;
                    }
                }
                __syncthreads();
            }

            // Write results back to global memory.
            {
                unsigned sharedIndex = startSharedIndex;
                unsigned globalIndex = startGlobalIndex;
                for (; globalIndex < endGlobalIndex; ++sharedIndex, ++globalIndex) {
                    data[globalIndex] = conflictFreeGet(sharedData, sharedIndex);
                }
            }
        }

        __global__ void addSums(unsigned n, int *__restrict__ data, const int *__restrict__ sums,
                                unsigned elementsPerThread) {
            unsigned startIndex = (blockIdx.x * blockDim.x + threadIdx.x) * elementsPerThread * 2U;
            unsigned endIndex = min(startIndex + elementsPerThread * 2U, n);
            int sum = sums[blockIdx.x];
            for (unsigned index = startIndex; index < endIndex; ++index) {
                data[index] += sum;
            }
        }

        void scanImpl(unsigned n, int *data, unsigned blockSize, unsigned elementsPerThread,
                      unsigned tileSize, unsigned sharedMemorySize, unsigned gridSize, int *sums) {
#define DISPATCH(N) scanPerBlock<N><<<gridSize, blockSize, sharedMemorySize>>>(n, data, sums);
            switch (elementsPerThread) {
            case 1U:
                DISPATCH(1U);
                break;
            case 2U:
                DISPATCH(2U);
                break;
            case 4U:
                DISPATCH(4U);
                break;
            case 8U:
                DISPATCH(8U);
                break;
            case 16U:
                DISPATCH(16U);
                break;
            default:
                printf("Error: elementsPerThread %u is not supported!\n", elementsPerThread);
                exit(1);
            }
#undef DISPATCH

            checkCUDAErrorFn("scanPerBlock kernel failed!");
            if (gridSize > 1U) {
                unsigned nextGridSize = (gridSize + tileSize - 1U) / tileSize;
                int *nextSums = nullptr;
                if (nextGridSize > 1U) {
                    nextSums = sums + gridSize;
                }
                scanImpl(gridSize, sums, blockSize, elementsPerThread, tileSize, sharedMemorySize,
                         nextGridSize, nextSums);
                addSums<<<gridSize, blockSize>>>(n, data, sums, elementsPerThread);
                checkCUDAErrorFn("addSums kernel failed!");
            }
        }

        void scan(int n, int *odata, const int *idata, const int blockSize,
                  const int elementsPerThread) {
            const auto dataSize = n * sizeof(int);

            int *dev_data;
            hipMalloc((void **)&dev_data, dataSize);
            checkCUDAErrorFn("hipMalloc dev_data failed!");
            hipMemcpy(dev_data, idata, dataSize, hipMemcpyHostToDevice);
            checkCUDAErrorFn("hipMemcpy to device failed!");

            // Allocate GPU memory for sums beforehand.
            unsigned tileSize = blockSize * elementsPerThread * 2U;
            unsigned sharedMemorySize = (conflictFreeIndex(tileSize - 1U) + 1U) * sizeof(int);
            unsigned gridSize = (n + tileSize - 1U) / tileSize;

            unsigned totalSumCount = 0U;
            {
                unsigned gridSize = n;
                while (true) {
                    gridSize = (gridSize + tileSize - 1U) / tileSize;
                    if (gridSize <= 1U) {
                        break;
                    }
                    totalSumCount += gridSize;
                }
            }
            int *dev_sums = nullptr;
            if (totalSumCount > 0U) {
                hipMalloc((void **)&dev_sums, totalSumCount * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_sums failed!");
            }

            hipDeviceSynchronize();
            timer().startGpuTimer();
            scanImpl(n, dev_data, blockSize, elementsPerThread, tileSize, sharedMemorySize,
                     gridSize, dev_sums);
            timer().endGpuTimer();

            hipMemcpy(odata, dev_data, dataSize, hipMemcpyDeviceToHost);
            checkCUDAErrorFn("hipMemcpy to host failed!");
            hipFree(dev_data);
            checkCUDAErrorFn("hipFree failed!");
            if (dev_sums != nullptr) {
                hipFree(dev_sums);
                checkCUDAErrorFn("hipFree failed!");
            }
        }
    }
}
